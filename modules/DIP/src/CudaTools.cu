#include "hip/hip_runtime.h"
// Copyright 2017 Caoyang Jiang

#include "Jcy/DIP/CudaTools.h"

JCY_WINDOWS_DISABLE_ALL_WARNING
#include <cstdio>
#include <iostream>
#include <string>
#include <vector>
#include "opencv2/core/core.hpp"
#include "opencv2/core/cuda.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/opencv.hpp"
#include "opencv2/photo/cuda.hpp"
#include "opencv2/xphoto.hpp"
JCY_WINDOWS_DISABLE_ALL_WARNING

namespace jcy
{
namespace cuda
{
__global__ static void AdjustContrastBrightnessKernel(
    uint8_t* data, int width, int height, double alpha, int beta)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((x < width) && (y < height))
  {
    for (int c = 0; c < 3; c++)
    {
      uint8_t* channel = data + width * height * c;
      uint32_t pix     = *(channel + (x + y * width));

      pix                = alpha * pix + beta;
      // if (pix > 255) pix = 255;

      pix = 255 * (pix >= 255) + pix * (pix < 255);

      // if(x==0)
      // {
      //   std::printf("thread 0: %d\n",pix);
      // }
      *(channel + (x + y * width)) = static_cast<uint8_t>(pix);
    }
  }
}

void Tools::AdjustContrastBrightness(cv::Mat& img, double alpha, int beta)
{
  int blockdimx, blockdimy, griddimx, griddimy;

  blockdimx = 32;
  blockdimy = 16;
  griddimx  = (img.size().width + blockdimx - 1) / blockdimx;
  griddimy  = (img.size().height + blockdimy - 1) / blockdimy;

  dim3 blks(griddimx, griddimy);
  dim3 threads(blockdimx, blockdimy);

  uint8_t* dpix = nullptr;
  hipMalloc(reinterpret_cast<void**>(&dpix),
             img.size().width * img.size().height * 3);
  hipMemcpy(dpix,
             img.data,
             img.size().width * img.size().height * 3,
             hipMemcpyHostToDevice);
  AdjustContrastBrightnessKernel<<<blks, threads>>>(
      dpix, img.size().width, img.size().height, alpha, beta);
  hipDeviceSynchronize();
  hipMemcpy(img.data,
             dpix,
             img.size().width * img.size().height * 3,
             hipMemcpyDeviceToHost);
  hipFree(dpix);
}

}  // namespace cuda
}  // namespace jcy
