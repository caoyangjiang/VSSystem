#include "hip/hip_runtime.h"
// Copyright 2017 Caoyang Jiang

#include "Jcy/DIP/CudaTools.h"

JCY_WINDOWS_DISABLE_ALL_WARNING
#include <iostream>
#include <string>
#include <vector>
#include "opencv2/core/core.hpp"
#include "opencv2/core/cuda.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/opencv.hpp"
#include "opencv2/photo/cuda.hpp"
#include "opencv2/xphoto.hpp"
JCY_WINDOWS_DISABLE_ALL_WARNING

namespace jcy
{
namespace cuda
{
__global__ static void CUDAKernelAddOneToVector(int* data)
{
  const int x  = blockIdx.x * blockDim.x + threadIdx.x;
  const int y  = blockIdx.y * blockDim.y + threadIdx.y;
  const int mx = gridDim.x * blockDim.x;

  data[y * mx + x] = data[y * mx + x] + 1;
}

void Tools::AdjustContrastBrightness(cv::Mat& img, double alpha, int beta)
{
  for (int y = 0; y < img.rows; y++)
  {
    for (int x = 0; x < img.cols; x++)
    {
      for (int c = 0; c < 3; c++)
      {
        img.at<cv::Vec3b>(y, x)[c] = cv::saturate_cast<uchar>(
            alpha * (img.at<cv::Vec3b>(y, x)[c]) + beta);
      }
    }
  }
}

}  // namespace cuda
}  // namespace jcy
