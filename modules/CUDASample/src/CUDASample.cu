#include "hip/hip_runtime.h"
// Copyright 2015 Jason Juang

#include "Hvr/CUDASample/CUDASample.cuh"

#include <vector>

#include "Hvr/CUDASample/CUDAConfig.h"

__global__ static void CUDAKernelAddOneToVector(int *data)
{
  const int x  = blockIdx.x * blockDim.x + threadIdx.x;
  const int y  = blockIdx.y * blockDim.y + threadIdx.y;
  const int mx = gridDim.x * blockDim.x;

  data[y * mx + x] = data[y * mx + x] + 1;
}

namespace hvr
{
CUDASample::CUDASample()
{
}

CUDASample::~CUDASample()
{
}

void CUDASample::AddOneToVector(std::vector<int> &in) const
{
  if (in.empty()) return;

  CUDAConfig cudaconfig;
  cudaconfig.w_a = static_cast<int>(in.size()) / cudaconfig.blk_h;
  cudaconfig.h_a = cudaconfig.blk_h;

  int *data;
  hipMallocManaged(reinterpret_cast<void **>(&data),
                    in.size() * sizeof(int),
                    hipMemAttachGlobal);

  for (int i = 0; i < static_cast<int>(in.size()); i++)
  {
    data[i] = in.at(i);
  }

  dim3 blks((cudaconfig.w_a >> cudaconfig.shift_w),
            (cudaconfig.h_a >> cudaconfig.shift_h));
  dim3 threads(cudaconfig.blk_w, cudaconfig.blk_h);

  CUDAKernelAddOneToVector<<<blks, threads>>>(data);

  hipDeviceSynchronize();

  for (int i = 0; i < static_cast<int>(in.size()); i++)
  {
    in.at(i) = data[i];
  }

  hipFree(data);
}

}  // namespace hvr
